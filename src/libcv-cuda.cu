#include "hip/hip_runtime.h"
#include "libcv-cuda.h"

#include <string.h>

__global__
void downscale(int width, int height, uint8_t *dst, const uint8_t *src)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < width && j < height) {
        int x = 0;
        for(int ii = 0; ii < 2; ii++) {
            for(int jj = 0; jj < 2; jj ++) {
                x += src[(j * 2 + jj) * width * 2 + i * 2 + ii];
            }
        }
        dst[j * width + i] = x / 4;
    }
        
}

uint8_t* src0;
uint8_t* dst;
void cuda_init(int width, int height) {
    int size0 = width * height;
    hipMalloc(&src0, size0);
    int size = width * height / 4;
    hipMalloc(&dst, size);
}

void downscale_ref(uint8_t* dest, const uint8_t* src, const int width, const int height) {
    for(int i = 0; i < width / 2; i ++) {
        for(int j = 0; j < height / 2; j ++) {
            int x = 0;
            for(int ii = 0; ii < 2; ii++) {
                for(int jj = 0; jj < 2; jj ++) {
                    x += src[(j * 2 + jj) * width + i * 2 + ii];
                }
            }
            dest[j*width/2 + i] = x / 4;
        }
    }
}

void downscale_cuda(uint8_t* dest, const uint8_t* src, const int width, const int height) {
    // memcpy(dest, src, width * height / 4);
    int size0 = width * height;
    int size = width * height / 4;
    hipMemcpy(src0, src, size0, hipMemcpyHostToDevice);


    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(width/2 / threadsPerBlock.x + 1, height/2 / threadsPerBlock.y + 1);

    downscale<<<numBlocks, threadsPerBlock>>>(width/2, height/2, dst, src0);
    hipMemcpy(dest, dst, size, hipMemcpyDeviceToHost);
}

void cuda_deinit() {
    hipFree(dst);
    hipFree(src0);
}