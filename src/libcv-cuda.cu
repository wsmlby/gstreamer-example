#include "hip/hip_runtime.h"
#include "libcv-cuda.h"

#include <string.h>

__global__
void copy(int n, uint8_t *x, const uint8_t *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      x[i] = y[i];
}
uint8_t* src0;
uint8_t* dst;
void cuda_init(int width, int height) {
    int size0 = width * height;
    hipMalloc(&src0, size0);
    int size = width * height / 4;
    hipMalloc(&dst, size);
}

void downscale_cuda(uint8_t* dest, const uint8_t* src, const int width, const int height) {
    // memcpy(dest, src, width * height / 4);
    int size0 = width * height;
    int size = width * height / 4;
    // hipMemcpy(src0, src, size0, hipMemcpyHostToDevice);
    // copy<<<16, 16>>>(size, dst, src0);
    hipMemcpy(dest, dst, size, hipMemcpyDeviceToHost);
}

void cuda_deinit() {
    hipFree(dst);
    hipFree(src0);
}